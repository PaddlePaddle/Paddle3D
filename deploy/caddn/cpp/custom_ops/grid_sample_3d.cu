#include "hip/hip_runtime.h"
//   Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>

#include "grid_sample_3d.h"
#include "paddle/include/experimental/ext_all.h"

#define CHECK_INPUT_GPU(x) PD_CHECK(x.is_gpu(), #x " must be a GPU Tensor.")

static __forceinline__ __device__ bool InBounds3D(int64_t d, int64_t h,
                                                  int64_t w, int64_t D,
                                                  int64_t H, int64_t W) {
  return d >= 0 && d < D && h >= 0 && h < H && w >= 0 && w < W;
}

#define CUDA_KERNEL_LOOP_TYPE(i, n, index_type)                  \
  index_type _i_n_d_e_x = blockIdx.x * blockDim.x + threadIdx.x; \
  for (index_type i = _i_n_d_e_x; _i_n_d_e_x < (n);              \
       _i_n_d_e_x += blockDim.x * gridDim.x, i = _i_n_d_e_x)

#define CUDA_KERNEL_LOOP(i, n) CUDA_KERNEL_LOOP_TYPE(i, n, int)

template <typename T>
static __forceinline__ __device__ T Unnormalize(T coord, int size,
                                                bool align_corners) {
  if (align_corners) {
    return ((coord + 1.f) / 2) * (size - 1);
  } else {
    return ((coord + 1.f) * size - 1) / 2;
  }
}

template <typename T>
static __forceinline__ __device__ T ClipIndexes(T in, int max_value) {
  return min(static_cast<T>(max_value), max(in, static_cast<T>(0)));
}

template <typename T>
static __forceinline__ __device__ T ReflectIndexes(T in, int twice_low,
                                                   int twice_high) {
  if (twice_low == twice_high) {
    return static_cast<T>(0);
  }
  T min = static_cast<T>(twice_low) / 2;
  T span = static_cast<T>(twice_high - twice_low) / 2;
  in = fabs(in - min);
  T extra = fmod(in, span);
  int flips = static_cast<int>(floor(in / span));
  if (flips % 2 == 0) {
    return extra + min;
  } else {
    return span - extra + min;
  }
}

template <typename T>
static __forceinline__ __device__ T ComputePositions(T coord, int size,
                                                     PaddingMode padding_mode,
                                                     bool align_corners) {
  coord = Unnormalize<T>(coord, size, align_corners);
  if (padding_mode == PaddingMode::border) {
    coord = ClipIndexes(coord, size - 1);
  } else if (padding_mode == PaddingMode::reflect) {
    if (align_corners) {
      coord = ReflectIndexes(coord, 0, 2 * (size - 1));
    } else {
      coord = ReflectIndexes(coord, -1, 2 * size - 1);
    }
    coord = ClipIndexes(coord, size - 1);
  }
  return coord;
}

template <typename T, typename index_t>
__global__ void GridSample3DCudaKernel(
    const index_t nthreads, index_t out_c, index_t out_d, index_t out_h,
    index_t out_w, index_t in_d, index_t in_h, index_t in_w, const T* input,
    const T* grid, T* output, const Mode interpolation_mode,
    const PaddingMode padding_mode, bool align_corners) {
  // printf("size: %d, %d, %d, %d, %d, %d \n", out_c, out_d, out_w, out_h, in_d,
  // in_w);
  index_t inp_sW = 1;
  index_t inp_sH = in_w;
  index_t inp_sD = in_h * in_w;
  index_t inp_sC = in_d * inp_sD;
  index_t inp_sN = out_c * inp_sC;

  index_t grid_sCoor = 1;
  index_t grid_sW = 3;
  index_t grid_sH = out_w * grid_sW;
  index_t grid_sD = out_h * grid_sH;
  index_t grid_sN = out_d * grid_sD;

  index_t out_sW = 1;
  index_t out_sH = out_w;
  index_t out_sD = out_h * out_w;
  index_t out_sC = out_d * out_sD;
  index_t out_sN = out_c * out_sC;

  CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
    const index_t w = index % out_w;
    const index_t h = (index / out_w) % out_h;
    const index_t d = (index / (out_h * out_w)) % out_d;
    const index_t n = index / (out_d * out_h * out_w);
    const index_t grid_offset =
        n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;
    // get the corresponding input x, y, z co-ordinates from grid
    T ix = grid[grid_offset];
    T iy = grid[grid_offset + grid_sCoor];
    T iz = grid[grid_offset + 2 * grid_sCoor];
    ix = ComputePositions(ix, in_w, padding_mode, align_corners);
    iy = ComputePositions(iy, in_h, padding_mode, align_corners);
    iz = ComputePositions(iz, in_d, padding_mode, align_corners);
    // printf("ix: %f, iy: %f, iz: %f \n", ix, iy, iz);
    if (interpolation_mode == Mode::bilinear) {
      // get corner pixel values from (x, y, z)
      // for 4d, we used north-east-south-west
      // for 5d, we add top-bottom
      index_t ix_tnw = static_cast<index_t>(std::floor(ix));
      index_t iy_tnw = static_cast<index_t>(std::floor(iy));
      index_t iz_tnw = static_cast<index_t>(std::floor(iz));

      index_t ix_tne = ix_tnw + 1;
      index_t iy_tne = iy_tnw;
      index_t iz_tne = iz_tnw;

      index_t ix_tsw = ix_tnw;
      index_t iy_tsw = iy_tnw + 1;
      index_t iz_tsw = iz_tnw;

      index_t ix_tse = ix_tnw + 1;
      index_t iy_tse = iy_tnw + 1;
      index_t iz_tse = iz_tnw;

      index_t ix_bnw = ix_tnw;
      index_t iy_bnw = iy_tnw;
      index_t iz_bnw = iz_tnw + 1;

      index_t ix_bne = ix_tnw + 1;
      index_t iy_bne = iy_tnw;
      index_t iz_bne = iz_tnw + 1;

      index_t ix_bsw = ix_tnw;
      index_t iy_bsw = iy_tnw + 1;
      index_t iz_bsw = iz_tnw + 1;

      index_t ix_bse = ix_tnw + 1;
      index_t iy_bse = iy_tnw + 1;
      index_t iz_bse = iz_tnw + 1;

      // get surfaces to each neighbor:
      T tnw = (ix_bse - ix) * (iy_bse - iy) * (iz_bse - iz);
      T tne = (ix - ix_bsw) * (iy_bsw - iy) * (iz_bsw - iz);
      T tsw = (ix_bne - ix) * (iy - iy_bne) * (iz_bne - iz);
      T tse = (ix - ix_bnw) * (iy - iy_bnw) * (iz_bnw - iz);
      T bnw = (ix_tse - ix) * (iy_tse - iy) * (iz - iz_tse);
      T bne = (ix - ix_tsw) * (iy_tsw - iy) * (iz - iz_tsw);
      T bsw = (ix_tne - ix) * (iy - iy_tne) * (iz - iz_tne);
      T bse = (ix - ix_tnw) * (iy - iy_tnw) * (iz - iz_tnw);

      auto inp_ptr_NC = input + n * inp_sN;
      auto out_ptr_NCDHW =
          output + n * out_sN + d * out_sD + h * out_sH + w * out_sW;
      for (index_t c = 0; c < out_c;
           ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
        *out_ptr_NCDHW = static_cast<T>(0);
        if (InBounds3D(iz_tnw, iy_tnw, ix_tnw, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW] *
              tnw;
        }
        if (InBounds3D(iz_tne, iy_tne, ix_tne, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW] *
              tne;
        }
        if (InBounds3D(iz_tsw, iy_tsw, ix_tsw, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW] *
              tsw;
        }
        if (InBounds3D(iz_tse, iy_tse, ix_tse, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW] *
              tse;
        }
        if (InBounds3D(iz_bnw, iy_bnw, ix_bnw, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW] *
              bnw;
        }
        if (InBounds3D(iz_bne, iy_bne, ix_bne, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW] *
              bne;
        }
        if (InBounds3D(iz_bsw, iy_bsw, ix_bsw, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW] *
              bsw;
        }
        if (InBounds3D(iz_bse, iy_bse, ix_bse, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW] *
              bse;
        }
      }
    } else if (interpolation_mode == Mode::nearest) {
      index_t ix_nearest = static_cast<index_t>(std::round(ix));
      index_t iy_nearest = static_cast<index_t>(std::round(iy));
      index_t iz_nearest = static_cast<index_t>(std::round(iz));

      // assign nearest neighor pixel value to output pixel
      auto inp_ptr_NC = input + n * inp_sN;
      auto out_ptr_NCDHW =
          output + n * out_sN + d * out_sD + h * out_sH + w * out_sW;
      for (index_t c = 0; c < out_c;
           ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
        if (InBounds3D(iz_nearest, iy_nearest, ix_nearest, in_d, in_h, in_w)) {
          *out_ptr_NCDHW =
              inp_ptr_NC[iz_nearest * inp_sD + iy_nearest * inp_sH +
                         ix_nearest * inp_sW];
        } else {
          *out_ptr_NCDHW = static_cast<T>(0);
        }
      }
    }
  }
}

std::vector<paddle::Tensor> GridSample3DCUDAForward(
    const paddle::Tensor& x, const paddle::Tensor& grid,
    const std::string& mode, const std::string& padding_mode,
    bool align_corners) {
  CHECK_INPUT_GPU(x);
  CHECK_INPUT_GPU(grid);
  PaddingMode enum_padding_mode;
  Mode enum_mode;
  if (padding_mode == "border") {
    enum_padding_mode = PaddingMode::border;
  } else if (padding_mode == "reflection") {
    enum_padding_mode = PaddingMode::reflect;
  } else {
    enum_padding_mode = PaddingMode::zeros;
  }

  if (mode == "nearest") {
    enum_mode = Mode::nearest;
  } else {
    enum_mode = Mode::bilinear;
  }
  const int n = grid.shape()[0];
  const int out_d = grid.shape()[1];
  const int out_h = grid.shape()[2];
  const int out_w = grid.shape()[3];
  const int c = x.shape()[1];
  const int in_d = x.shape()[2];
  const int in_h = x.shape()[3];
  const int in_w = x.shape()[4];

  auto output = paddle::full({n, c, out_d, out_h, out_w}, 0,
                             paddle::DataType::FLOAT32, paddle::GPUPlace());
  const int count = static_cast<int>(n * out_d * out_h * out_w);

  int max_threads_per_block = 512;
  int block_num = (count - 1) / max_threads_per_block + 1;
  // printf("size: %d, %d, %d, %d, %d, %d \n", n, c, out_d, out_h, count,
  // block_num);
  GridSample3DCudaKernel<float, int>
      <<<block_num, max_threads_per_block, 0, x.stream()>>>(
          count, c, out_d, out_h, out_w, in_d, in_h, in_w, x.data<float>(),
          grid.data<float>(), output.data<float>(), enum_mode,
          enum_padding_mode, align_corners);

  hipError_t error_check;
  error_check = hipGetLastError();
  if (error_check != hipSuccess) {
    printf("%s\n", hipGetErrorString(error_check));
  }
  // printf("size: %d, %d, %d, %d, %d, %d \n", n, c, out_d, out_h, count,
  // block_num);
  return {output};
}

template <typename T>
static __forceinline__ __device__ T UnnormalizeWithMask(T coord, int size,
                                                        bool align_corners,
                                                        T* grad_in) {
  if (align_corners) {
    *grad_in = static_cast<T>(size - 1) / 2;
    return ((coord + 1.f) / 2) * (size - 1);
  } else {
    *grad_in = static_cast<T>(size) / 2;
    return ((coord + 1.f) * size - 1) / 2;
  }
}

template <typename T>
static __forceinline__ __device__ T ClipIndexesWithMask(T in, int clip_limit,
                                                        T* grad_in) {
  if (in <= static_cast<T>(0)) {
    *grad_in = static_cast<T>(0);
    return static_cast<T>(0);
  } else {
    T max = static_cast<T>(clip_limit - 1);
    if (in >= max) {
      *grad_in = static_cast<T>(0);
      return max;
    } else {
      *grad_in = static_cast<T>(1);
      return in;
    }
  }
}

template <typename T>
static __forceinline__ __device__ T ReflectIndexesWithMask(T in, int twice_low,
                                                           int twice_high,
                                                           T* grad_in) {
  if (twice_low == twice_high) {
    *grad_in = static_cast<T>(0);
    return static_cast<T>(0);
  }
  int grad_in_mult_;
  T min = static_cast<T>(twice_low) / 2;
  T span = static_cast<T>(twice_high - twice_low) / 2;
  in = in - min;
  if (in < static_cast<T>(0)) {
    grad_in_mult_ = -1;
    in = -in;
  } else {
    grad_in_mult_ = 1;
  }
  T extra = fmod(in, span);
  int flips = static_cast<int>(floor(in / span));
  if (flips % 2 == 0) {
    *grad_in = static_cast<T>(grad_in_mult_);
    return extra + min;
  } else {
    *grad_in = static_cast<T>(-grad_in_mult_);
    return span - extra + min;
  }
}

template <typename T>
static __forceinline__ __device__ T
ComputePositionsWithMask(T coord, int size, PaddingMode padding_mode,
                         bool align_corners, T* grad_in) {
  T grad_clip, grad_refl;
  coord = UnnormalizeWithMask<T>(coord, size, align_corners, grad_in);
  if (padding_mode == PaddingMode::border) {
    coord = ClipIndexesWithMask(coord, size, &grad_clip);
    *grad_in = (*grad_in) * grad_clip;
  } else if (padding_mode == PaddingMode::reflect) {
    if (align_corners) {
      coord = ReflectIndexesWithMask(coord, 0, 2 * (size - 1), &grad_refl);
    } else {
      coord = ReflectIndexesWithMask(coord, -1, 2 * size - 1, &grad_refl);
    }
    coord = ClipIndexesWithMask(coord, size, &grad_clip);
    *grad_in = (*grad_in) * grad_refl * grad_clip;
  }

  return coord;
}

template <typename T>
static __forceinline__ __device__ void AtomicAdd3D(
    T* data, int64_t d, int64_t h, int64_t w, int64_t sD, int64_t sH,
    int64_t sW, int64_t D, int64_t H, int64_t W, T delta) {
  if (InBounds3D(d, h, w, D, H, W)) {
    atomicAdd(data + d * sD + h * sH + w * sW, delta);
  }
}

template <typename T, typename index_t>
__global__ void GridSample3DCudaBackwardKernel(
    const index_t nthreads, const T* grad_output, const T* input, const T* grid,
    index_t out_c, index_t out_d, index_t out_h, index_t out_w, index_t in_d,
    index_t in_h, index_t in_w, T* grad_input, T* grad_grid, const Mode mode,
    const PaddingMode padding_mode, bool align_corners) {
  index_t inp_sW = 1;
  index_t inp_sH = in_w;
  index_t inp_sD = in_h * in_w;
  index_t inp_sC = in_d * inp_sD;
  index_t inp_sN = out_c * inp_sC;

  index_t grid_sCoor = 1;
  index_t grid_sW = 3;
  index_t grid_sH = out_w * grid_sW;
  index_t grid_sD = out_h * grid_sH;
  index_t grid_sN = out_d * grid_sD;

  index_t gOut_sW = 1;
  index_t gOut_sH = out_w;
  index_t gOut_sD = out_h * out_w;
  index_t gOut_sC = out_d * gOut_sD;
  index_t gOut_sN = out_c * gOut_sC;

  CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
    const index_t w = index % out_w;
    const index_t h = (index / out_w) % out_h;
    const index_t d = (index / (out_h * out_w)) % out_d;
    const index_t n = index / (out_d * out_h * out_w);
    const auto grid_offset =
        n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

    // get the corresponding input x, y, z co-ordinates from grid
    T ix = grid[grid_offset];
    T iy = grid[grid_offset + grid_sCoor];
    T iz = grid[grid_offset + 2 * grid_sCoor];

    // multipliers for gradients on ix, iy, and iz
    T gix_mult, giy_mult, giz_mult;
    ix = ComputePositionsWithMask(ix, in_w, padding_mode, align_corners,
                                  &gix_mult);
    iy = ComputePositionsWithMask(iy, in_h, padding_mode, align_corners,
                                  &giy_mult);
    iz = ComputePositionsWithMask(iz, in_d, padding_mode, align_corners,
                                  &giz_mult);

    if (mode == Mode::bilinear) {
      // get corner pixel values from (x, y, z)
      // for 4d, we used north-east-south-west
      // for 5d, we add top-bottom
      index_t ix_tnw = static_cast<index_t>(std::floor(ix));
      index_t iy_tnw = static_cast<index_t>(std::floor(iy));
      index_t iz_tnw = static_cast<index_t>(std::floor(iz));

      index_t ix_tne = ix_tnw + 1;
      index_t iy_tne = iy_tnw;
      index_t iz_tne = iz_tnw;

      index_t ix_tsw = ix_tnw;
      index_t iy_tsw = iy_tnw + 1;
      index_t iz_tsw = iz_tnw;

      index_t ix_tse = ix_tnw + 1;
      index_t iy_tse = iy_tnw + 1;
      index_t iz_tse = iz_tnw;

      index_t ix_bnw = ix_tnw;
      index_t iy_bnw = iy_tnw;
      index_t iz_bnw = iz_tnw + 1;

      index_t ix_bne = ix_tnw + 1;
      index_t iy_bne = iy_tnw;
      index_t iz_bne = iz_tnw + 1;

      index_t ix_bsw = ix_tnw;
      index_t iy_bsw = iy_tnw + 1;
      index_t iz_bsw = iz_tnw + 1;

      index_t ix_bse = ix_tnw + 1;
      index_t iy_bse = iy_tnw + 1;
      index_t iz_bse = iz_tnw + 1;

      // get surfaces to each neighbor:
      T tnw = (ix_bse - ix) * (iy_bse - iy) * (iz_bse - iz);
      T tne = (ix - ix_bsw) * (iy_bsw - iy) * (iz_bsw - iz);
      T tsw = (ix_bne - ix) * (iy - iy_bne) * (iz_bne - iz);
      T tse = (ix - ix_bnw) * (iy - iy_bnw) * (iz_bnw - iz);
      T bnw = (ix_tse - ix) * (iy_tse - iy) * (iz - iz_tse);
      T bne = (ix - ix_tsw) * (iy_tsw - iy) * (iz - iz_tsw);
      T bsw = (ix_tne - ix) * (iy - iy_tne) * (iz - iz_tne);
      T bse = (ix - ix_tnw) * (iy - iy_tnw) * (iz - iz_tnw);

      T gix = static_cast<T>(0), giy = static_cast<T>(0),
        giz = static_cast<T>(0);
      index_t gOut_offset =
          n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
      index_t inp_offset_NC = n * inp_sN;
      T* gInp_ptr_NC = grad_input + n * inp_sN;
      for (index_t c = 0; c < out_c; ++c, gOut_offset += gOut_sC,
                   gInp_ptr_NC += inp_sC, inp_offset_NC += inp_sC) {
        T gOut = grad_output[gOut_offset];

        AtomicAdd3D(gInp_ptr_NC, iz_tnw, iy_tnw, ix_tnw, inp_sD, inp_sH, inp_sW,
                    in_d, in_h, in_w, tnw * gOut);
        AtomicAdd3D(gInp_ptr_NC, iz_tne, iy_tne, ix_tne, inp_sD, inp_sH, inp_sW,
                    in_d, in_h, in_w, tne * gOut);
        AtomicAdd3D(gInp_ptr_NC, iz_tsw, iy_tsw, ix_tsw, inp_sD, inp_sH, inp_sW,
                    in_d, in_h, in_w, tsw * gOut);
        AtomicAdd3D(gInp_ptr_NC, iz_tse, iy_tse, ix_tse, inp_sD, inp_sH, inp_sW,
                    in_d, in_h, in_w, tse * gOut);
        AtomicAdd3D(gInp_ptr_NC, iz_bnw, iy_bnw, ix_bnw, inp_sD, inp_sH, inp_sW,
                    in_d, in_h, in_w, bnw * gOut);
        AtomicAdd3D(gInp_ptr_NC, iz_bne, iy_bne, ix_bne, inp_sD, inp_sH, inp_sW,
                    in_d, in_h, in_w, bne * gOut);
        AtomicAdd3D(gInp_ptr_NC, iz_bsw, iy_bsw, ix_bsw, inp_sD, inp_sH, inp_sW,
                    in_d, in_h, in_w, bsw * gOut);
        AtomicAdd3D(gInp_ptr_NC, iz_bse, iy_bse, ix_bse, inp_sD, inp_sH, inp_sW,
                    in_d, in_h, in_w, bse * gOut);

        // calculate grad_grid
        if (InBounds3D(iz_tnw, iy_tnw, ix_tnw, in_d, in_h, in_w)) {
          T tnw_val = input[inp_offset_NC + iz_tnw * inp_sD + iy_tnw * inp_sH +
                            ix_tnw * inp_sW];
          gix -= tnw_val * (iy_bse - iy) * (iz_bse - iz) * gOut;
          giy -= tnw_val * (ix_bse - ix) * (iz_bse - iz) * gOut;
          giz -= tnw_val * (ix_bse - ix) * (iy_bse - iy) * gOut;
        }
        if (InBounds3D(iz_tne, iy_tne, ix_tne, in_d, in_h, in_w)) {
          T tne_val = input[inp_offset_NC + iz_tne * inp_sD + iy_tne * inp_sH +
                            ix_tne * inp_sW];
          gix += tne_val * (iy_bsw - iy) * (iz_bsw - iz) * gOut;
          giy -= tne_val * (ix - ix_bsw) * (iz_bsw - iz) * gOut;
          giz -= tne_val * (ix - ix_bsw) * (iy_bsw - iy) * gOut;
        }
        if (InBounds3D(iz_tsw, iy_tsw, ix_tsw, in_d, in_h, in_w)) {
          T tsw_val = input[inp_offset_NC + iz_tsw * inp_sD + iy_tsw * inp_sH +
                            ix_tsw * inp_sW];
          gix -= tsw_val * (iy - iy_bne) * (iz_bne - iz) * gOut;
          giy += tsw_val * (ix_bne - ix) * (iz_bne - iz) * gOut;
          giz -= tsw_val * (ix_bne - ix) * (iy - iy_bne) * gOut;
        }
        if (InBounds3D(iz_tse, iy_tse, ix_tse, in_d, in_h, in_w)) {
          T tse_val = input[inp_offset_NC + iz_tse * inp_sD + iy_tse * inp_sH +
                            ix_tse * inp_sW];
          gix += tse_val * (iy - iy_bnw) * (iz_bnw - iz) * gOut;
          giy += tse_val * (ix - ix_bnw) * (iz_bnw - iz) * gOut;
          giz -= tse_val * (ix - ix_bnw) * (iy - iy_bnw) * gOut;
        }
        if (InBounds3D(iz_bnw, iy_bnw, ix_bnw, in_d, in_h, in_w)) {
          T bnw_val = input[inp_offset_NC + iz_bnw * inp_sD + iy_bnw * inp_sH +
                            ix_bnw * inp_sW];
          gix -= bnw_val * (iy_tse - iy) * (iz - iz_tse) * gOut;
          giy -= bnw_val * (ix_tse - ix) * (iz - iz_tse) * gOut;
          giz += bnw_val * (ix_tse - ix) * (iy_tse - iy) * gOut;
        }
        if (InBounds3D(iz_bne, iy_bne, ix_bne, in_d, in_h, in_w)) {
          T bne_val = input[inp_offset_NC + iz_bne * inp_sD + iy_bne * inp_sH +
                            ix_bne * inp_sW];
          gix += bne_val * (iy_tsw - iy) * (iz - iz_tsw) * gOut;
          giy -= bne_val * (ix - ix_tsw) * (iz - iz_tsw) * gOut;
          giz += bne_val * (ix - ix_tsw) * (iy_tsw - iy) * gOut;
        }
        if (InBounds3D(iz_bsw, iy_bsw, ix_bsw, in_d, in_h, in_w)) {
          T bsw_val = input[inp_offset_NC + iz_bsw * inp_sD + iy_bsw * inp_sH +
                            ix_bsw * inp_sW];
          gix -= bsw_val * (iy - iy_tne) * (iz - iz_tne) * gOut;
          giy += bsw_val * (ix_tne - ix) * (iz - iz_tne) * gOut;
          giz += bsw_val * (ix_tne - ix) * (iy - iy_tne) * gOut;
        }
        if (InBounds3D(iz_bse, iy_bse, ix_bse, in_d, in_h, in_w)) {
          T bse_val = input[inp_offset_NC + iz_bse * inp_sD + iy_bse * inp_sH +
                            ix_bse * inp_sW];
          gix += bse_val * (iy - iy_tnw) * (iz - iz_tnw) * gOut;
          giy += bse_val * (ix - ix_tnw) * (iz - iz_tnw) * gOut;
          giz += bse_val * (ix - ix_tnw) * (iy - iy_tnw) * gOut;
        }
      }
      if (grad_grid != nullptr) {
        T* gGrid_ptr_NDHW = grad_grid + index * grid_sW;
        gGrid_ptr_NDHW[0] = gix_mult * gix;
        gGrid_ptr_NDHW[1] = giy_mult * giy;
        gGrid_ptr_NDHW[2] = giz_mult * giz;
      }
    } else if (mode == Mode::nearest) {
      auto ix_nearest = static_cast<index_t>(std::round(ix));
      auto iy_nearest = static_cast<index_t>(std::round(iy));
      auto iz_nearest = static_cast<index_t>(std::round(iz));

      // assign nearest neighor pixel value to output pixel
      index_t gOut_offset =
          n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
      T* gInp_ptr_NC = grad_input + n * inp_sN;
      for (index_t c = 0; c < out_c;
           ++c, gOut_offset += gOut_sC, gInp_ptr_NC += inp_sC) {
        AtomicAdd3D(gInp_ptr_NC, iz_nearest, iy_nearest, ix_nearest, inp_sD,
                    inp_sH, inp_sW, in_d, in_h, in_w, grad_output[gOut_offset]);
      }
      if (grad_grid != nullptr) {
        T* gGrid_ptr_NDHW = grad_grid + index * grid_sW;
        gGrid_ptr_NDHW[0] = static_cast<T>(0);
        gGrid_ptr_NDHW[1] = static_cast<T>(0);
        gGrid_ptr_NDHW[2] = static_cast<T>(0);
      }
    }
  }
}

std::vector<paddle::Tensor> GridSample3DCUDABackward(
    const paddle::Tensor& x, const paddle::Tensor& grid,
    const paddle::Tensor& grad_out, const std::string& mode,
    const std::string& padding_mode, bool align_corners) {
  PaddingMode enum_padding_mode;
  Mode enum_mode;
  if (padding_mode == "border") {
    enum_padding_mode = PaddingMode::border;
  } else if (padding_mode == "reflection") {
    enum_padding_mode = PaddingMode::reflect;
  } else {
    enum_padding_mode = PaddingMode::zeros;
  }

  if (mode == "nearest") {
    enum_mode = Mode::nearest;
  } else {
    enum_mode = Mode::bilinear;
  }

  const int out_d = grid.shape()[1];
  const int out_h = grid.shape()[2];
  const int out_w = grid.shape()[3];
  const int n = x.shape()[0];
  const int c = x.shape()[1];
  const int in_d = x.shape()[2];
  const int in_h = x.shape()[3];
  const int in_w = x.shape()[4];

  auto grid_grad_output =
      paddle::empty({n, out_d, out_h, out_w, 3}, paddle::DataType::FLOAT32,
                    paddle::GPUPlace());
  auto x_grad_output =
      paddle::full({n, c, in_d, in_h, in_w}, 0, paddle::DataType::FLOAT32,
                   paddle::GPUPlace());

  const int count = static_cast<int>(n * out_d * out_h * out_w);

  int max_threads_per_block = 512;
  int block_num = (count - 1) / max_threads_per_block + 1;

  GridSample3DCudaBackwardKernel<float, int>
      <<<block_num, max_threads_per_block, 0, x.stream()>>>(
          count, grad_out.data<float>(), x.data<float>(), grid.data<float>(), c,
          out_d, out_h, out_w, in_d, in_h, in_w, x_grad_output.data<float>(),
          grid_grad_output.data<float>(), enum_mode, enum_padding_mode,
          align_corners);

  return {x_grad_output};
}
