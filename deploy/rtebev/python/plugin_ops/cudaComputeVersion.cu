
#include <hip/hip_runtime.h>
#include <cstdio>

int main() {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int v = prop.major * 10 + prop.minor;
  printf("%d", v);
  return 0;
}