#include "hip/hip_runtime.h"
#include <paddle/extension.h>

#include <vector>

#define BLOCK 512

template <typename data_t>
__global__ void cv_undistort_cuda_kernel(const data_t* xy_coords,
                                         const data_t* distortion_coeffs,
                                         data_t* xy_undist, float eps,
                                         int max_iterations, int input_numel) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;

  float k1, k2, k3, k4, p1, p2;
  k1 = distortion_coeffs[0];
  k2 = distortion_coeffs[1];

  k3 = distortion_coeffs[2];
  k4 = distortion_coeffs[3];

  p1 = distortion_coeffs[4];
  p2 = distortion_coeffs[5];

  input_numel = (input_numel) / 2;
  for (int i = gid; i < input_numel; i += blockDim.x * gridDim.x) {
    int xy_number = i * 2;
    float x = xy_coords[xy_number];
    float y = xy_coords[xy_number + 1];

    for (int iteration = 0; iteration < max_iterations; iteration++) {
      float xd, yd;
      xd = xy_coords[xy_number];
      yd = xy_coords[xy_number + 1];
      float fx, fy, fx_x, fx_y, fy_x, fy_y, r, d, d_r, d_x, d_y;

      r = x * x + y * y;
      d = 1.0 + r * (k1 + r * (k2 + r * (k3 + r * k4)));

      fx = d * x + 2 * p1 * x * y + p2 * (r + 2 * x * x) - xd;
      fy = d * y + 2 * p2 * x * y + p1 * (r + 2 * y * y) - yd;

      //# Compute derivative of d over [x, y]
      d_r = k1 + r * (2.0 * k2 + r * (3.0 * k3 + r * 4.0 * k4));
      d_x = 2.0 * x * d_r;
      d_y = 2.0 * y * d_r;

      //# Compute derivative of fx over x and y.
      fx_x = d + d_x * x + 2.0 * p1 * y + 6.0 * p2 * x;
      fx_y = d_y * x + 2.0 * p1 * x + 2.0 * p2 * y;

      //# Compute derivative of fy over x and y.
      fy_x = d_x * y + 2.0 * p2 * y + 2.0 * p1 * x;
      fy_y = d + d_y * y + 2.0 * p2 * x + 6.0 * p1 * y;

      //
      float denominator, x_numerator, y_numerator, step_x, step_y;
      denominator = fy_x * fx_y - fx_x * fy_y;
      x_numerator = fx * fy_y - fy * fx_y;
      y_numerator = fy * fx_x - fx * fy_x;

      if (abs(denominator) > eps) {
        step_x = x_numerator / denominator;
        step_y = y_numerator / denominator;
      } else {
        step_x = 0.0;
        step_y = 0.0;
      }
      x += step_x;
      y += step_y;
    }
    xy_undist[xy_number] = x;
    xy_undist[xy_number + 1] = y;
  }
}

std::vector<paddle::Tensor> cv_undistort_cuda(
    const paddle::Tensor& xy_coords,
    const paddle::Tensor& input_distortion_coeffs, int max_iterations,
    float eps) {
  auto xy_undist = paddle::Tensor(paddle::PlaceType::kGPU, xy_coords.shape());

  PD_CHECK(input_distortion_coeffs.size() == 6,
           "PD_CHECK returns input_distortion_coeffs.size() <=6.");

  int input_numel = xy_coords.size();
  int grid = (input_numel + BLOCK - 1) / BLOCK;

  PD_DISPATCH_FLOATING_TYPES(
      xy_coords.type(), "cv_undistort_cuda_kernel", ([&] {
        cv_undistort_cuda_kernel<data_t>
            <<<grid, BLOCK, 0, xy_coords.stream()>>>(
                xy_coords.data<data_t>(),
                input_distortion_coeffs.data<data_t>(),
                xy_undist.data<data_t>(), eps, max_iterations, input_numel);
      }));

  return {xy_undist};
}
