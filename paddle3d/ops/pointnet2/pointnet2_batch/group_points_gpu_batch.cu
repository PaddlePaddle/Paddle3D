#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

/*
batch version of point grouping, modified from the original implementation of
official PointNet++ codes. Written by Shaoshuai Shi All Rights Reserved 2018.
*/

#include "paddle/extension.h"

#define THREADS_PER_BLOCK 512
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

__global__ void group_points_cuda_kernel_batch(const int b, const int c,
                                               const int n, const int npoints,
                                               const int nsample,
                                               const float *__restrict__ points,
                                               const int *__restrict__ idx,
                                               float *__restrict__ out) {
  // points: (B, C, N)
  // idx: (B, npoints, nsample)
  // output:
  //      out: (B, C, npoints, nsample)
  int bs_idx = blockIdx.z;
  int c_idx = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int pt_idx = index / nsample;
  if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

  int sample_idx = index % nsample;

  idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx;
  int in_idx = bs_idx * c * n + c_idx * n + idx[0];
  int out_idx = bs_idx * c * npoints * nsample + c_idx * npoints * nsample +
                pt_idx * nsample + sample_idx;

  out[out_idx] = points[in_idx];
}

void group_points_cuda_launcher_batch(const int b, const int c, const int n,
                                      const int npoints, const int nsample,
                                      const float *points, const int *idx,
                                      float *out) {
  // points: (B, C, N)
  // idx: (B, npoints, nsample)
  // output:
  //      out: (B, C, npoints, nsample)
  hipError_t err;
  dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c,
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  group_points_cuda_kernel_batch<<<blocks, threads>>>(b, c, n, npoints, nsample,
                                                      points, idx, out);
  // hipDeviceSynchronize();  // for using printf in kernel function
  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}

__global__ void group_points_grad_cuda_kernel_batch(
    const int b, const int c, const int n, const int npoints, const int nsample,
    const float *__restrict__ grad_out, const int *__restrict__ idx,
    float *__restrict__ grad_points) {
  // grad_out: (B, C, npoints, nsample)
  // idx: (B, npoints, nsample)
  // output:
  //      grad_points: (B, C, N)
  int bs_idx = blockIdx.z;
  int c_idx = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int pt_idx = index / nsample;
  if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

  int sample_idx = index % nsample;
  grad_out += bs_idx * c * npoints * nsample + c_idx * npoints * nsample +
              pt_idx * nsample + sample_idx;
  idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx;

  atomicAdd(grad_points + bs_idx * c * n + c_idx * n + idx[0], grad_out[0]);
}

void group_points_grad_cuda_launcher_batch(const int b, const int c,
                                           const int n, const int npoints,
                                           const int nsample,
                                           const float *grad_out,
                                           const int *idx, float *grad_points) {
  // grad_out: (B, C, npoints, nsample)
  // idx: (B, npoints, nsample)
  // output:
  //      grad_points: (B, C, N)
  hipError_t err;
  dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c,
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  group_points_grad_cuda_kernel_batch<<<blocks, threads>>>(
      b, c, n, npoints, nsample, grad_out, idx, grad_points);

  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}
